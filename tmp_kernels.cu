#include "hip/hip_runtime.h"
#include <vector>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <cmath>

// CUDA �J�[�l��
__global__ void temporal_denoise_kernel(uint8_t** frames, int numFrames,
    int center, uint8_t* out,
    int w, int h, int stride,
    float alphaLow, float alphaMid, float alphaHigh,
    float strength)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= w || y >= h) return;

    int idx = y * stride + x;

    // ���ԕ���
    float sum = 0.0f;
    for (int i = 0; i < numFrames; i++) {
        sum += frames[i][idx];
    }
    float avg = sum / numFrames;
    float cur = frames[center][idx];

    // ���g���ʂ̏d�ݕt��
    float low = (1.0f - alphaLow) * cur + alphaLow * avg;
    float mid = (1.0f - alphaMid) * cur + alphaMid * avg;
    float high = (1.0f - alphaHigh) * cur + alphaHigh * avg;

    float denoised = (low + mid + high) / 3.0f;

    // ����Ƃ̃u�����h
    float outVal = (1.0f - strength) * cur + strength * denoised;

    out[idx] = (uint8_t)fminf(fmaxf(outVal, 0.0f), 255.0f);
}

extern "C" void runTemporalDenoise(
    const std::vector<const uint8_t*>& srcFrames,
    const std::vector<int>& strides,
    uint8_t* dst, int w, int h, int dstStride,
    int radius,
    float alphaLow, float alphaMid, float alphaHigh,
    float strength)
{
    int numFrames = (int)srcFrames.size();
    int center = radius;

    size_t size = h * dstStride;

    // GPU �������m��
    std::vector<uint8_t*> d_frames(numFrames);
    for (int i = 0; i < numFrames; i++) {
        hipMalloc(&d_frames[i], size);
        hipMemcpy(d_frames[i], srcFrames[i], size, hipMemcpyHostToDevice);
    }

    uint8_t* d_out;
    hipMalloc(&d_out, size);

    // �t���[���|�C���^�z��� GPU �ɓ]��
    uint8_t** d_frame_ptrs;
    hipMalloc(&d_frame_ptrs, numFrames * sizeof(uint8_t*));
    hipMemcpy(d_frame_ptrs, d_frames.data(), numFrames * sizeof(uint8_t*), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((w + block.x - 1) / block.x,
        (h + block.y - 1) / block.y);

    // �J�[�l���Ăяo��
    temporal_denoise_kernel << <grid, block >> > (d_frame_ptrs, numFrames, center,
        d_out, w, h, dstStride,
        alphaLow, alphaMid, alphaHigh,
        strength);

    hipMemcpy(dst, d_out, size, hipMemcpyDeviceToHost);

    // ��n��
    for (int i = 0; i < numFrames; i++) hipFree(d_frames[i]);
    hipFree(d_out);
    hipFree(d_frame_ptrs);
}
